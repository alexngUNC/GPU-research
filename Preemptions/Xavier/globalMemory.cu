#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include "testbench.h"
#define PERCENTAGE_SHARED (9000.0)


// Kernel for computing aX + Y where a is a scalar and X, Y are vectors
__global__ void dirty(int n, float a, float *x, float *y, float *z1, float *z2) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Index for loop
  int index = (int) 12.0*PERCENTAGE_SHARED*((double) threadIdx.x);

  // Infinite loop
  while (1) {
    y[i] = a * x[i] + y[i];

    // Copy result to a range of 12 * percentage shared
    int limit = (int) 12.0*PERCENTAGE_SHARED;
    for (int j=0; j<limit; j++) {
      z1[index+j] = y[i];   
      z2[index+j] = y[i];   
      z2[index+j] += z1[index+j];
    }
    __syncthreads();
  }
}


int main() {
  // Needed for the SAFE macro
  hipError_t err;

  // Host pointers
  float *h_x, *h_y, *h_z1, *h_z2;

  // Length of vectors (number of total threads) 
  int n = 16384;

  // Allocate CPU memory and initialize vectors
  h_x = (float*) malloc(n * sizeof(float));
  h_y = (float*) malloc(n * sizeof(float));
  h_z1 = (float*) malloc(n * 12 * PERCENTAGE_SHARED * sizeof(float));
  h_z2 = (float*) malloc(n * 12 * PERCENTAGE_SHARED * sizeof(float));
  for (int i=1; i<=n; i++) {
    h_x[i-1] = i;
    h_y[i-1] = i+1;
  }

  // Pointers to vectors stored on the GPU
  float *d_x, *d_y, *d_z1, *d_z2;

  // Allocate memory on the GPU for vectors
  SAFE(hipMalloc(&d_x, n * sizeof(float)));
  SAFE(hipMalloc(&d_y, n * sizeof(float)));
  SAFE(hipMalloc(&d_z1, n * 12 * PERCENTAGE_SHARED * sizeof(float)));
  SAFE(hipMalloc(&d_z2, n * 12 * PERCENTAGE_SHARED * sizeof(float)));

  // Copy memory from CPU to GPU
  SAFE(hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice));
  SAFE(hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice));
  SAFE(hipMemcpy(d_z1, h_z1, n * sizeof(float), hipMemcpyHostToDevice));
  SAFE(hipMemcpy(d_z2, h_z2, n * sizeof(float), hipMemcpyHostToDevice));

  // Define a
  float a = 10.0;
  
  // Partition the L1 cache to avoid allocating shared memory
  int carveout = 0;
  SAFE(hipFuncSetAttribute(reinterpret_cast<const void*>(dirty), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

  // Execute the kernel - 2 TBs per SM for all SMs
  dirty<<<16, 1024>>>(n, a, d_x, d_y, d_z1, d_z2);
  
  // Copy the memory from the GPU back to the CPU
  SAFE(hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost));

  // Free the GPU memory
  SAFE(hipFree(d_x));
  SAFE(hipFree(d_y));

  // Free the CPU memory
  free(h_x);
  free(h_y);
  return 0;
}
