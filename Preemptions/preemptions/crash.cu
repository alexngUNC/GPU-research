#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "testbench.h"

__global__ void dereference(int *a) {
	a[0] = 1;
}

int
main() {
	// init driver api
	SAFE_D(hipInit(0));

	// create a context on the gpu
	hipDevice_t device;
	SAFE_D(hipDeviceGet(&device, 0));
	hipCtx_t context;
	SAFE_D(hipCtxCreate(&context, 0, device));

	// dereference memory address
	dereference<<<1, 1>>>((int*) NULL);

	// sync for error
	SAFE(hipDeviceSynchronize());
	return 0;
}
