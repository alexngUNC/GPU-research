#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"
#define CONCURRENT_TB 38
#define PERCENTAGE_SHARED (2.0/2.0)
#define SHARED_MEM_TB 49152


__global__ void vecAdd(float *a, float *b, int n, int *flag) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// dynamic shared memory for result
	extern __shared__ float res[];

	// index for loop
	int idx = (int) 12.0 * PERCENTAGE_SHARED * ((double) threadIdx.x);

	// loop to fill up shared memory
	b[i] += a[i];
	int limit = (int) 12.0 * PERCENTAGE_SHARED;
	for (int j=0; j<limit; j++) {
		res[idx + j] = b[i];
	}
	__syncthreads();
	// tell CPU that shared memory is fully saturated
	*flag = 0;
	// spin with desired shared memory usage
	while (1) {}
}


int
main()
{
	// vector length
	int n = CONCURRENT_TB * 1024;
	// host memory
	float *h_a, *h_b;
	h_a = (float *) malloc(n * sizeof(float));
	h_b = (float *) malloc(n * sizeof(float));
	for (int i=0; i<n; i++) {
		h_a[i] = 1;
		h_b[i] = 2;
	}

	// device memory
	float *d_a, *d_b;
	int bytes = n * sizeof(float);
	SAFE(hipMalloc(&d_a, bytes));
	SAFE(hipMalloc(&d_b, bytes));
	SAFE(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
	SAFE(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

	// flag for synchronization
	int *flag;
	SAFE(hipHostAlloc(&flag, sizeof(int), hipHostMallocMapped));
	*flag = 1;

	// launch kernel
	int sharedMem = SHARED_MEM_TB * PERCENTAGE_SHARED;
	vecAdd<<<CONCURRENT_TB, 1024, sharedMem>>>(d_a, d_b, n, flag);
	while (*flag) {}
	printf("Shared memory is fully saturated!\n");
	SAFE(hipDeviceSynchronize());

	// print result
	SAFE(hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost));
	for (int i=0; i<10; i++) {
		printf("%f\t", h_b[i]);
	}
	printf("\n");

	// free memory
	SAFE(hipFree(d_a));
	SAFE(hipFree(d_b));
	free(h_a);
	free(h_b);
	return 0;
}
