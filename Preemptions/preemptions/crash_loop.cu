#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "testbench.h"
#define PAGES 100000
#define PAGE_SIZE 65536

__global__ void dereference(int *a) {
	a[threadIdx.x] = 1;
}

int
main() {
	// init driver api
	SAFE_D(hipInit(0));

	// create a context on the gpu
	hipDevice_t device;
	SAFE_D(hipDeviceGet(&device, 0));
	hipCtx_t context;
	SAFE_D(hipCtxCreate(&context, 0, device));

	// allocate device memory
	hipDeviceptr_t d_a[PAGES];
	for (int i=0; i<PAGES; i++) {
		SAFE_D(hipMalloc(&d_a[i], PAGE_SIZE));
	}

	for (int i=0; i<PAGES; i++) {
		// dereference memory address
		dereference<<<1, 1024>>>((int*) NULL);
	}

	// sync to catch an error
	PRINT_ERROR(hipDeviceSynchronize());
	int *d_test;
	PRINT_ERROR(hipMalloc(&d_test, 10*4));
	for (int i=0; i<PAGES; i++) {
		hipFree(d_a[i]);
	}
	return 0;
}
