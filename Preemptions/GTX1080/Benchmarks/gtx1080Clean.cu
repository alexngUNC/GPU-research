#include "hip/hip_runtime.h"
#include <stdio.h>
#include "testbench.h"

__global__ void spin() {
    while (1) {}
}

int main()  {
    hipError_t err;
    spin<<<40, 1024>>>();
    SAFE(hipDeviceSynchronize());
    printf("Exited\n");
    return 0;
}
